#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
# define PI           3.14159265358979323846

using namespace std ;

# define DELLEXPORT extern "C" __declspec(dllexport)

__global__ void cudaMinJerkKernel(float *intRampFn, float *rampFn, float * amp,float * Damp, float * freq,  float * Dfreq,
  float * phase, float * Dphase, float * sum, unsigned n_vectors, unsigned arr_size){
  
  unsigned idx = threadIdx.x+blockDim.x*blockIdx.x;
  if (idx < arr_size){
    float temp = 0;
    double sampleRate = 400;
    double dt = 1/sampleRate;
    for (unsigned i = 0; i < n_vectors; i++)
      temp += (Damp[i]*rampFn[idx]+amp[i])*sinf((double)(Dfreq[i]*intRampFn[idx] + freq[i]*(idx+1)*dt +
       Dphase[i]*rampFn[idx] + phase[i]));

    sum[idx] = temp;}
}

  
  

DELLEXPORT void cudaInitialize(){
  
  hipFree(0);

}


DELLEXPORT void cudaRearrange(float *intRampFn, float *rampFn, float * amp, float * Damp, float * freq,  float * Dfreq,
  float * phase, float * Dphase, float * output, int arr_size, int n_vector){

  const int nTPB = 1024;

  const long long int ARRAY_BYTES = arr_size * sizeof(float) ;
  const long long int Param_BYTES = n_vector * sizeof(float) ;

  float *amp_d, *freq_d, *phase_d, *d_out ,*d_intRamp, *d_Ramp,*Damp_d, *Dfreq_d, *Dphase_d;

  hipMalloc((void **) &amp_d, Param_BYTES) ;
  hipMalloc((void **) &freq_d, Param_BYTES) ;
  hipMalloc((void **) &phase_d, Param_BYTES) ;
  hipMalloc((void **) &Damp_d, Param_BYTES) ;
  hipMalloc((void **) &Dfreq_d, Param_BYTES) ;
  hipMalloc((void **) &Dphase_d, Param_BYTES) ;

  hipMalloc((void **) &d_out, ARRAY_BYTES) ;
  hipMalloc((void **) &d_intRamp, ARRAY_BYTES) ;
  hipMalloc((void **) &d_Ramp, ARRAY_BYTES) ;

  hipMemcpy(amp_d, amp, Param_BYTES, hipMemcpyHostToDevice) ;
  hipMemcpy(freq_d, freq, Param_BYTES, hipMemcpyHostToDevice) ;
  hipMemcpy(phase_d, phase, Param_BYTES, hipMemcpyHostToDevice) ;
  hipMemcpy(Damp_d, Damp, Param_BYTES, hipMemcpyHostToDevice) ;
  hipMemcpy(Dfreq_d, Dfreq, Param_BYTES, hipMemcpyHostToDevice) ;
  hipMemcpy(Dphase_d, Dphase, Param_BYTES, hipMemcpyHostToDevice) ;

  hipMemcpy(d_intRamp, intRampFn, ARRAY_BYTES, hipMemcpyHostToDevice) ;
  hipMemcpy(d_Ramp, rampFn, ARRAY_BYTES, hipMemcpyHostToDevice) ;

  cudaMinJerkKernel<<<(arr_size + nTPB -1 )/nTPB,nTPB>>>(d_intRamp, d_Ramp, amp_d, Damp_d, freq_d, Dfreq_d, phase_d, Dphase_d,
    d_out,n_vector,arr_size) ;

  hipMemcpy(output, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost) ;

  hipFree(amp_d);
  hipFree(freq_d);
  hipFree(phase_d);
  hipFree(Damp_d);
  hipFree(Dfreq_d);
  hipFree(Dphase_d);

  hipFree(d_Ramp);
  hipFree(d_intRamp);
  hipFree(d_out);
}



